// 2D Convolution in CUDA


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define MASK_DIM 7
#define MASK_OFFSET (MASK_DIM / 2)

__constant__ int mask[MASK_DIM * MASK_DIM];

__global__ void conv2d(int* matrix, int* result, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int start_r = row - MASK_OFFSET;
    int start_c = col - MASK_OFFSET;
    
    int temp = 0;
    
    for (int i = 0; i < MASK_DIM; i++) {
        for (int j = 0; j < MASK_DIM; j++) {
            if ((start_r + i) >= 0 && (start_r + i) < N) {
                if ((start_c + j) >= 0 && (start_c + j) < N) {
                    temp += matrix[(start_r + i) * N + (start_c + j)] * mask[i * MASK_DIM + j];
                }
            }
        }
    }
    result[row * N + col] = temp;
}

void init_matrix(int* matrix, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            matrix[i * N + j] = rand() % 100;
        }
    }
}

void verify_result(int* matrix, int* mask, int* result, int N) {
    int temp, offset_r, offset_c;
    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            temp = 0;
            for (int k = 0; k < MASK_DIM; k++) {
                offset_r = i - MASK_OFFSET + k;
                
                for (int l = 0; l < MASK_DIM; l++) {
                    offset_c = j - MASK_OFFSET + l;
                    
                    if (offset_r >= 0 && offset_r < N) {
                        if (offset_c >= 0 && offset_c < N) {
                            temp += matrix[offset_r * N + offset_c] * mask[k * MASK_DIM + l];
                        }
                    }
                }
            }
            assert(result[i * N + j] == temp);
        }
    }
}

int main() {
    int N = 1 << 10;
    size_t bytes_n = N * N * sizeof(int);
    
    int* matrix = new int[N * N];
    int* result = new int[N * N];
    init_matrix(matrix, N);
    
    size_t bytes_m = MASK_DIM * MASK_DIM * sizeof(int);
    
    int* h_mask = new int[MASK_DIM * MASK_DIM];
    init_matrix(h_mask, MASK_DIM);
    
    int* d_matrix, *d_result;
    hipMalloc(&d_matrix, bytes_n);
    hipMalloc(&d_result, bytes_n);
    
    hipMemcpy(d_matrix, matrix, bytes_n, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);
    
    int THREADS = 16;
    int BLOCKS = (N + THREADS - 1) / THREADS;
    
    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(BLOCKS, BLOCKS);
    
    conv2d <<< grid_dim, block_dim >>> (d_matrix, d_result, N);
    
    hipMemcpy(result, d_result, bytes_n, hipMemcpyDeviceToHost);
    
    verify_result(matrix, h_mask, result, N);
    printf("SUCCESS!!!");
    
    delete[] matrix; delete[] result; delete[] h_mask;
    hipFree(d_matrix); hipFree(d_result);
    return 0;
}
